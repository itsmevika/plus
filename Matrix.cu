#include <iostream>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
 
#define BLOCK_DIM 2 //размер субматрицы
int M, K;
 
using namespace std;
 
__global__ void matrixAdd (int *A, int *B, int *C, int M, int K) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
 
    int index = col * M + row;
 
    //сложение на GPU
    if (col < M && row < K) { 
        C[index] = A[index] + B[index];
    }
}
 
int main() {
 
    cout << "M: ";
    cin >> M;
    cout << "K: ";
    cin >> K;
 
    int *A = new int [M*K];
 
 
    int *B = new int [M*K];
 
 
    int *C = new int [M*K];
 
 
    //заполнение матриц
    for(int i=0; i<M; i++)
        for (int j=0; j<K; j++){
            A[i*M+j] = 2;
            B[i*M+j] = 1;
            C[i*M+j] = 0;
        }
 
    int *dev_a, *dev_b, *dev_c; //указатели на выделяемую память
 
    int size = M * K * sizeof(int); //выделяемая память
 
    hipMalloc((void**)&dev_a, size); //выделение памяти
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);
 
    hipMemcpy(dev_a, A, size, hipMemcpyHostToDevice); //копирование на GPU
    hipMemcpy(dev_b, B, size, hipMemcpyHostToDevice);
 
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM); //число выделенных блоков
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x, (K+dimBlock.y-1)/dimBlock.y); //размер и размерность сетки
    printf("dimGrid.x = %d, dimGrid.y = %d\n", dimGrid.x, dimGrid.y); //выводится размер сетки
 
    matrixAdd<<<dimGrid,dimBlock>>>(dev_a, dev_b, dev_c, M, K); //вызов ядра
    hipDeviceSynchronize(); 
    
    hipMemcpy(C, dev_c, size, hipMemcpyDeviceToHost);
 
    //вывод    результата
    printf("Result Matrix C:\n");
    for(int i=0; i<M; i++){ 
        for (int j=0; j<K; j++){
            printf("%d\t", C[i] );
        }
        printf("\n");
    }
 
 
    hipFree(dev_a); //освобождение памяти
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
